#include "hip/hip_runtime.h"
#include <stdio.h>
// Do not alter the preprocessor directives
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <cstdlib>


#define NUM_CHANNELS 1

#define BLOCKSIZE 512

#define MinVal(x, y) (((x) < (y)) ? (x) : (y))
#define MaxVal(x, y) (((x) > (y)) ? (x) : (y))



__global__ void minKernel(uint8_t *image, int size, uint8_t *odata)
{
	extern __shared__ volatile uint8_t sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(BLOCKSIZE)+tid;
	unsigned int gridSize = BLOCKSIZE * gridDim.x;

	uint8_t val = image[i];
	i += gridSize;
	while (i < size) {
		val = MinVal(image[i], val);
		i += gridSize;
	}
	sdata[tid] = val;
	__syncthreads();

	// Parallel reduction v5
	for (int i = (tid + 32); ((tid < 32) && (i < BLOCKSIZE)); i += 32)
		sdata[tid] = MinVal(sdata[tid], sdata[i]);

	if (tid < 16) sdata[tid] = MinVal(sdata[tid], sdata[tid + 16]);
	if (tid < 8)  sdata[tid] = MinVal(sdata[tid], sdata[tid + 8]);
	if (tid < 4)  sdata[tid] = MinVal(sdata[tid], sdata[tid + 4]);
	if (tid < 2)  sdata[tid] = MinVal(sdata[tid], sdata[tid + 2]);
	if (tid == 0) odata[blockIdx.x] = MinVal(sdata[tid], sdata[tid + 1]);
}



__global__ void maxKernel(uint8_t *image, int size, uint8_t *odata)
{
	extern __shared__ volatile uint8_t sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(BLOCKSIZE)+tid;
	unsigned int gridSize = BLOCKSIZE * gridDim.x;

	uint8_t val = image[i];
	i += gridSize;
	while (i < size) {
		val = MaxVal(image[i], val);
		i += gridSize;
	}
	sdata[tid] = val;
	__syncthreads();


	// Parallel reduction v5

	for (int i = (tid + 32); ((tid < 32) && (i < BLOCKSIZE)); i += 32)
		sdata[tid] = MaxVal(sdata[tid], sdata[i]);

	if (tid < 16) sdata[tid] = MaxVal(sdata[tid], sdata[tid + 16]);
	if (tid < 8)  sdata[tid] = MaxVal(sdata[tid], sdata[tid + 8]);
	if (tid < 4)  sdata[tid] = MaxVal(sdata[tid], sdata[tid + 4]);
	if (tid < 2)  sdata[tid] = MaxVal(sdata[tid], sdata[tid + 2]);
	if (tid == 0) odata[blockIdx.x] = MaxVal(sdata[tid], sdata[tid + 1]);
}

__global__ void SubKernel(uint8_t *image, uint8_t min_value)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	image[i] = image[i] - min_value;
}

__global__ void ScaleKernel(uint8_t *image, float scale_constant)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	image[i] = image[i] * scale_constant;
}

int main() {

	int width; //image width
	int height; //image height
	int bpp;  //bytes per pixel if the image was RGB (not used)



	// Load a grayscale bmp image to an unsigned integer array with its height and weight.
	//  (uint8_t is an alias for "unsigned char")
	uint8_t* image = stbi_load("./samples/1280x843.bmp", &width, &height, &bpp, NUM_CHANNELS);
	size_t image_size = width * height * sizeof(uint8_t);

	// Print for sanity check
	printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
	printf("Height: %d \n", height);
	printf("Width: %d \n", width);


	//Start Counter
	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//Device variables
	uint8_t* min_d;
	uint8_t* max_d;
	uint8_t* image_d;

	//Kernel execution sizes
	unsigned int GRIDSIZE = width / (BLOCKSIZE * 1.0);
	unsigned int GRIDSIZE2 = ceil((width * height) / (BLOCKSIZE * 1.0));

	//Allocate memory for host varables
	uint8_t* min_host = (uint8_t*)malloc(GRIDSIZE * sizeof(uint8_t));
	uint8_t* max_host = (uint8_t*)malloc(GRIDSIZE * sizeof(uint8_t));

	//CUDA allocate memory
	hipMalloc((void**)&min_d, sizeof(uint8_t) * GRIDSIZE);
	hipMalloc((void**)&max_d, sizeof(uint8_t) * GRIDSIZE);
	hipMalloc((void**)&image_d, image_size);

	//Copy image values to device
	hipMemcpy(image_d, image, image_size, hipMemcpyHostToDevice);



	//kernels to find minimum and maximum values
	minKernel << <GRIDSIZE, BLOCKSIZE, sizeof(uint8_t)*BLOCKSIZE >> > (image_d, width* height, min_d);
	maxKernel << <GRIDSIZE, BLOCKSIZE, sizeof(uint8_t)*BLOCKSIZE >> > (image_d, width*height, max_d);


	//Get min and max values
	hipMemcpy(min_host, min_d, sizeof(uint8_t) * GRIDSIZE, hipMemcpyDeviceToHost);
	hipMemcpy(max_host, max_d, sizeof(uint8_t) * GRIDSIZE, hipMemcpyDeviceToHost);


	//Subtraction Kernel
	SubKernel << <GRIDSIZE2, BLOCKSIZE >> > (image_d, min_host[0]);


	float scale_constant = 255.0f / (max_host[0] - min_host[0]);

	//Scale Kernel
	ScaleKernel << <GRIDSIZE2, BLOCKSIZE >> > (image_d, scale_constant);

	//Copy image from device to host
	hipMemcpy(image, image_d, image_size, hipMemcpyDeviceToHost);


	//Stop timing
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("\nTime to calculate results(GPU Time): %f ms.\n\n", elapsed_time_ms);

	printf("Minimum Pixel Value: %d\n", min_host[0]);
	printf("Maximum Pixel Value: %d\n", max_host[0]);

	// Write image array into a bmp file
	stbi_write_bmp("./samples/out_img.bmp", width, height, 1, image);
	printf("\nEnchanced image successfully saved.\n");




	//free memory
	free(min_host);
	free(max_host);
	hipFree(min_d);
	hipFree(max_d);
	hipFree(image_d);

	return 0;
}
